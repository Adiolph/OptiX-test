#include "hip/hip_runtime.h"
#include <optix.h>

using namespace optix;

struct PerRayData_pathtrace
{
  unsigned int hitID;
  unsigned int seed;
};

rtDeclareVariable(float3, hit_pos, attribute hit_pos);
rtDeclareVariable(PerRayData_pathtrace, prd_radiance, rtPayload, );

RT_PROGRAM void closest_hit_radiance()
{
  PerRayData_pathtrace.hitID = hit_pos.x / 10;
};


