#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <common.h>
#include <helpers.h>
#include <random.h>  // OptiX random header file in SDK/cuda/random.h

using namespace optix;

struct PerRayData_pathtrace
{
  unsigned int hitID;
  unsigned int seed;
};

rtDeclareVariable(float3, source_pos, , );  // position of source
rtDeclareVariable(rtObject, top_object, , );  // group object
rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim, rtLaunchDim, );
rtBuffer<uint, 1> output_id;  // record the id of dom that photon hit, 0 if no hit


RT_PROGRAM void point_source()
{
  float3 ray_origin = source_pos;
  PerRayData_pathtrace prd;
  prd.hitID = 0;
  prd.seed = tea<4>(launch_index, 0);
  float cos_th = 2.0 * rnd(prd.seed) - 1.0;
  float sin_th = sqrt(1 - cos_th*cos_th);
  float cos_ph = cos(2 * pi * rnd(prd.seed));
  float sin_ph = sin(2 * pi * rnd(prd.seed));
  float3 ray_direction = make_float3(cos_th*cos_ph, cos_th*sin_ph, sin_th);
  Ray ray = make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, 0.01, RT_DEFAULT_MAX);
  rtTrace(top_object, ray, prd);
  output_id[launch_index] = prd.hitID;
}
